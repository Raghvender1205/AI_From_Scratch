#include "hip/hip_runtime.h"
#define USE_MNIST_LOADER
#define MNIST_DOUBLE
#include "mnist.h"
#include "layer.h"

#include <hip/hip_runtime.h>
#include <cstdio>
#include <time.h>

static mnist_data *train_set, *test_set;
static unsigned int train_cnt, test_cnt;

// Define layers of CNN
static Layer l_input = Layer(0, 0, 28*28);
static Layer l_c1 = Layer(5*5, 6, 24*24*6);
static Layer l_s1 = Layer(4*4, 1, 6*6*6);
static Layer l_f = Layer(6*6*6, 10, 10);

static void learn();
static unsigned int classify(double data[28][28]);
static void test();
static double forward_pass(double data[28][28]);
static double back_pass();

static inline void loaddata() {
    mnist_load("data/train-images.idx3-ubyte", "data/train-labels.idx1-ubyte",
		&train_set, &train_cnt);
	mnist_load("data/t10k-images.idx3-ubyte", "data/t10k-labels.idx1-ubyte",
		&test_set, &test_cnt);
}

int main(int argc, const char **argv) {
    srand(time(NULL));

    hipError_t err = hipInit(0);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA initialization failed with error code - %d\n", err);
        return 1;
    }

    loaddata();
    learn();
    test();

    return 0;
}

// Forward Propogation of a single row in a dataset
static double forward_pass(double data[28][28]) {
    float input[28][28];

    for (int i = 0; i < 28; i++) {
        for (int j = 0; j < 28; ++j) {
            input[i][j] = data[i][j];
        }
    }

    l_input.clear();
    l_c1.clear();
	l_s1.clear();
	l_f.clear();

    clock_t start, end;
    start = clock();

    l_input.setOutput((float*) input);

    fp_preact_c1<<<64, 64>>>((float (*)[28])l_input.output, (float (*)[24][24])l_c1.preact, (float (*)[5][5])l_c1.weight);
	fp_bias_c1<<<64, 64>>>((float (*)[24][24])l_c1.preact, l_c1.bias);
	apply_step_function<<<64, 64>>>(l_c1.preact, l_c1.output, l_c1.O);

	fp_preact_s1<<<64, 64>>>((float (*)[24][24])l_c1.output, (float (*)[6][6])l_s1.preact, (float (*)[4][4])l_s1.weight);
	fp_bias_s1<<<64, 64>>>((float (*)[6][6])l_s1.preact, l_s1.bias);
	apply_step_function<<<64, 64>>>(l_s1.preact, l_s1.output, l_s1.O);

	fp_preact_f<<<64, 64>>>((float (*)[6][6])l_s1.output, l_f.preact, (float (*)[6][6][6])l_f.weight);
	fp_bias_f<<<64, 64>>>(l_f.preact, l_f.bias);
	apply_step_function<<<64, 64>>>(l_f.preact, l_f.output, l_f.O);
	
	end = clock();
	return ((double) (end - start)) / CLOCKS_PER_SEC;
}

// Back propogation to update weights
static double back_pass() {
    clock_t start, end;

    start = clock();

    bp_weight_f<<<64, 64>>>((float (*)[6][6][6])l_f.d_weight, l_f.d_preact, (float (*)[6][6])l_s1.output);
	bp_bias_f<<<64, 64>>>(l_f.bias, l_f.d_preact);

	bp_output_s1<<<64, 64>>>((float (*)[6][6])l_s1.d_output, (float (*)[6][6][6])l_f.weight, l_f.d_preact);
	bp_preact_s1<<<64, 64>>>((float (*)[6][6])l_s1.d_preact, (float (*)[6][6])l_s1.d_output, (float (*)[6][6])l_s1.preact);
	bp_weight_s1<<<64, 64>>>((float (*)[4][4])l_s1.d_weight, (float (*)[6][6])l_s1.d_preact, (float (*)[24][24])l_c1.output);
	bp_bias_s1<<<64, 64>>>(l_s1.bias, (float (*)[6][6])l_s1.d_preact);

	bp_output_c1<<<64, 64>>>((float (*)[24][24])l_c1.d_output, (float (*)[4][4])l_s1.weight, (float (*)[6][6])l_s1.d_preact);
	bp_preact_c1<<<64, 64>>>((float (*)[24][24])l_c1.d_preact, (float (*)[24][24])l_c1.d_output, (float (*)[24][24])l_c1.preact);
	bp_weight_c1<<<64, 64>>>((float (*)[5][5])l_c1.d_weight, (float (*)[24][24])l_c1.d_preact, (float (*)[28])l_input.output);
	bp_bias_c1<<<64, 64>>>(l_c1.bias, (float (*)[24][24])l_c1.d_preact);


	apply_grad<<<64, 64>>>(l_f.weight, l_f.d_weight, l_f.M * l_f.N);
	apply_grad<<<64, 64>>>(l_s1.weight, l_s1.d_weight, l_s1.M * l_s1.N);
	apply_grad<<<64, 64>>>(l_c1.weight, l_c1.d_weight, l_c1.M * l_c1.N);

	end = clock();
	return ((double) (end - start)) / CLOCKS_PER_SEC;
}

// Unfold the input layer
static void unfold_input(double input[28][28], double unfolded[24*24][5*5]) {
    int a = 0;
    (void)unfold_input;

    for (int i = 0; i < 2; ++i) {
        for (int j = 0; j < 2; ++j) {
            int b = 0;
            for (int x = i; x < i + 2; ++x) {
                for (int y = j; y < j + 2; ++y) {
                    unfolded[a][b++] = input[x][y];
                }
            }
            a++;
        }
    }
}

static void learn() {
    static hipblasHandle_t blas;
    hipblasHandle_t(&blas);

    float err;
    int iter = 50;

    double time_taken = 0.0;
    fprintf(stdout, "Learning\n");

    while (iter < 0 || iter-- > 0) {
        err = 0.0f;

        for (int i = 0; i < train_cnt; ++i) {
            float tmp_err;

            time_taken += forward_pass(train_set[i].data);

            l_f.bp_clear();
			l_s1.bp_clear();
			l_c1.bp_clear();

			// Euclid distance of train_set[i]
			makeError<<<10, 1>>>(l_f.d_preact, l_f.output, train_set[i].label, 10);
			hipblasSnrm2(blas, 10, l_f.d_preact, 1, &tmp_err);
			err += tmp_err;

			time_taken += back_pass();
        }
        
        err /= train_cnt;
        fprintf(stdout, "Error: %e, time_on_gpu: %lf\n", err, time_taken);

        if (err < threshold) {
            fprintf(stdout, "Training completed, error less than threshold\n");
            break;
        }
    }

    fprintf(stdout, "\n Time - %lf\n", time_taken);
}

// Returns label of given data (0-9)
static unsigned int classify(double data[28][28]) {
    float res[10];

	forward_pass(data);

	unsigned int max = 0;

	hipMemcpy(res, l_f.output, sizeof(float) * 10, hipMemcpyDeviceToHost);

	for (int i = 1; i < 10; ++i) {
		if (res[max] < res[i]) {
			max = i;
		}
	}

	return max;
}

// Perform forward propogation of test data
static void test() {
    int error = 0;

    for (int i = 0; i < test_cnt; ++i) {
        if (classify(test_set[i].data) != test_set[i].label) {
            ++error;
        }
    }

    fprintf(stdout, "Error Rate: %.2lf%%\n", 
            double(error) / double(test_cnt) * 100.0);
}