#include <hipDNN.h>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <vector>
#include <utility>

#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }


int main(int argc, const char* argv[]) {


    int gpu_id = 0;
    int img_ht = 2048;
    int img_wd = 2048;


    hipSetDevice(gpu_id);

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                        /*format=*/HIPDNN_TENSOR_NHWC,
                                        /*dataType=*/HIPDNN_DATA_FLOAT,
                                        /*batch_size=*/1,
                                        /*channels=*/1,
                                        /*image_height=*/img_ht,
                                        /*image_width=*/img_wd));

    hipdnnFilterDescriptor_t kernel_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                                        /*dataType=*/HIPDNN_DATA_FLOAT,
                                        /*format=*/HIPDNN_TENSOR_NCHW,
                                        /*out_channels=*/1,
                                        /*in_channels=*/1,
                                        /*kernel_height=*/3,
                                        /*kernel_width=*/3));

    hipdnnConvolutionDescriptor_t convolution_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                             /*pad_height=*/0,
                                             /*pad_width=*/0,
                                             /*vertical_stride=*/1,
                                             /*horizontal_stride=*/1,
                                             /*dilation_height=*/1,
                                             /*dilation_width=*/1,
                                             /*mode=*/HIPDNN_CONVOLUTION,
                                             /*computeType=*/HIPDNN_DATA_FLOAT));

    int batch_size{0}, channels{0}, height{0}, width{0};
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor,
                                                   input_descriptor,
                                                   kernel_descriptor,
                                                   &batch_size,
                                                   &channels,
                                                   &height,
                                                   &width));




    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                                        /*format=*/HIPDNN_TENSOR_NHWC,
                                        /*dataType=*/HIPDNN_DATA_FLOAT,
                                        /*batch_size=*/1,
                                        /*channels=*/1,
                                        /*image_height=*/height,
                                        /*image_width=*/width));

    hipdnnConvolutionFwdAlgo_t convolution_algorithm;
    checkCUDNN(
      hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                          input_descriptor,
                                          kernel_descriptor,
                                          convolution_descriptor,
                                          output_descriptor,
                                          HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                          /*memoryLimitInBytes=*/0,
                                          &convolution_algorithm));

    size_t workspace_bytes;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                     input_descriptor,
                                                     kernel_descriptor,
                                                     convolution_descriptor,
                                                     output_descriptor,
                                                     convolution_algorithm,
                                                     &workspace_bytes));
    std::cerr << "Workspace size: " <<  workspace_bytes  << "bytes"
            << std::endl;


    void* d_workspace{nullptr};
    hipMalloc(&d_workspace, workspace_bytes);

    int image_dims = img_ht * img_wd;
    int image_bytes = image_dims * sizeof(float);
    float *h_input = new float[image_bytes];
    for(int i=0; i< image_dims; i++){
        h_input[i] = 1;
    }

    float* d_input{nullptr};
    hipMalloc(&d_input, image_bytes);
    hipMemcpy(d_input, h_input, image_bytes, hipMemcpyHostToDevice);

    float* d_output{nullptr};
    hipMalloc(&d_output, image_bytes);
    hipMemset(d_output, 0, image_bytes);

    // clang-format off
    const float kernel_template[3][3] = {
    {0.5, 0.5, 0.5},
    {0.5, 0.5, 0.5},
    {0.5, 0.5, 0.5}
    };
    // clang-format on

    float h_kernel[1][1][3][3];
    for (int kernel = 0; kernel < 1; ++kernel) {
    for (int channel = 0; channel < 1; ++channel) {
      for (int row = 0; row < 3; ++row) {
        for (int column = 0; column < 3; ++column) {
          h_kernel[kernel][channel][row][column] = kernel_template[row][column];
        }
      }
    }
    }

    float* d_kernel{nullptr};
    hipMalloc(&d_kernel, sizeof(h_kernel));
    hipMemcpy(d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);

    const float alpha = 1.0f, beta = 0.0f;

    checkCUDNN(hipdnnConvolutionForward(cudnn,
                                     &alpha,
                                     input_descriptor,
                                     d_input,
                                     kernel_descriptor,
                                     d_kernel,
                                     convolution_descriptor,
                                     convolution_algorithm,
                                     d_workspace,
                                     workspace_bytes,
                                     &beta,
                                     output_descriptor,
                                     d_output));



    float* h_output = new float[image_bytes];
    hipMemcpy(h_output, d_output, image_bytes, hipMemcpyDeviceToHost);


    std::vector<std::pair<int,int> > miss;

    for(int i=0; i<height; i++){
        for(int j=0; j<width; j++){
            //std::cout<<h_output[i*height +j]<<" ";
            if(h_output[i*height +j] != 4.5){
                miss.push_back(std::make_pair(i,j));
            }
        }
        //std::cout<<"\n";
    }

    std::cout<<miss.size()<<"\n";
    for(int i=0;i<miss.size();i++){
        std::cout<<miss[i].first<<","<<miss[i].second<<"\n";
    }

    delete[] h_output;
    hipFree(d_kernel);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_workspace);

    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

    hipdnnDestroy(cudnn);
}