#include "hip/hip_runtime.h"
#include "bce_cost.hh"
#include "nn_exception.hh"

#include <math.h>
#include <iostream>
#include <assert.h>

__global__ void  binaryCrossEntropyCost(float* predictions, float* target, int size, float* cost) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        float partial_cost = target[index] * logf(predictions[index]) + 
                                + (1.0f - target[index]) * logf(1.0f - predictions[index]);

        atomicAdd(cost, -partial_cost / size);
    }
}

__global__ void dBinaryCrossEntropyCost(float* predictions, float* target, float* dY, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index < size) {
        dY[size] = -1.0 * (target[index] / predictions[index] - (1 - target[index]) / (1 - predictions[index]));
    }
}

float BCECost::cost(Matrix predictions, Matrix target) {
    assert(predictions.shape.x == target.shape.x);

    float* cost;
    hipMallocManaged(&cost, sizeof(float));
    *cost = 0.0f;

    dim3 block_size(256);
    dim3 num_of_blocks((predictions.shape.x + block_size.x - 1) / block_size.x);
    binaryCrossEntropyCost<<<num_of_blocks, block_size>>>(predictions.data_device.get(), 
                                                        target.data_device.get(), predictions.shape.x, cost);
    
    hipDeviceSynchronize();
    NNException::throwIfDeviceErrorsOccurred("Cannot compute binary cross entropy cost");

    float cost_value = *cost;
    hipFree(cost);

    return cost_value;
}

Matrix BCECost::dCost(Matrix predictions, Matrix target, Matrix dY) {
    assert(predictions.shape.x == target.shape.x);

	dim3 block_size(256);
	dim3 num_of_blocks((predictions.shape.x + block_size.x - 1) / block_size.x);
    dBinaryCrossEntropyCost<<<num_of_blocks, block_size>>>(predictions.data_device.get(), 
                                                        target.data_device.get(), dY.data_device.get(), predictions.shape.x);

    NNException::throwIfDeviceErrorsOccurred("Cannot compute derivative for binary cross entropy");
    return dY;
}