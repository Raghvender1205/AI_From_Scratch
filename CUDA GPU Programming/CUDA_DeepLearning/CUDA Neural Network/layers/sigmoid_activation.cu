#include "hip/hip_runtime.h"
#include "sigmoid_activation.hh"
#include "../nn_utils/nn_exception.hh"
#include <iostream>

__device__ float sigmoid(float x) {
    return 1.0f / (1 + exp(-x));
}

__global__ void sigmoidActivationForward(float* Z, float* A, int Z_x_dim, int Z_y_dim) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < Z_x_dim * Z_y_dim) {
        A[index] = sigmoid(Z[index]);
    }
}

__global__ void sigmoidActivationBackward(float* Z, float* dA, float* dZ, int Z_x_dim, int Z_y_dim) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < Z_x_dim * Z_y_dim) {
        dZ[index] = dA[index] * sigmoid(Z[index]) * (1 - sigmoid(Z[index]));
    }
}

SigmoidActivation::SigmoidActivation(std::string name) {
    this->name = name;
}

SigmoidActivation::~SigmoidActivation() { }

Matrix& SigmoidActivation::forward(Matrix& Z) {
    this->Z = Z;
    A.allocateMemoryIfNotAllocated(Z.shape);

    dim3 block_size(256);
    dim3 num_of_blocks((Z.shape.y * Z.shape.x + block_size.x - 1) / block_size.x);

    sigmoidActivationForward<<<num_of_blocks, block_size>>>(Z.data_device.get(), A.data_device.get(), Z.shape.x, Z.shape.y);

    NNException::throwIfDeviceErrorOccurred("Cannot perform sigmoid forward propogation");

    return A;
}

Matrix& SigmoidActivation::backward(Matrix& dA, float learning_rate) {
    dZ.allocateMemoryIfNotAllocated(Z.shape);

    dim3 block_size(256);
    dim3 num_of_blocks((Z.shape * Z.shape.x + block_size.x - 1) / block_size.x);
    sigmoidActivationBackward<<<num_of_blocks, block_size>>>(Z.data_device.get(), dA.data_device.get(), dZ.data_device.get(), Z.shape.x, Z.shape.y);

    NNException::throwIfDeviceErrorsOccurred("Cannot perform sigmoid back propogation");
    return dZ;
}
