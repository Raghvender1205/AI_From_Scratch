#include "neural_network.hh"
#include "nn_utils/nn_exception.hh"

NeuralNetwork::NeuralNetwork(float learning_rate) : learning_rate(learning_rate) { }

NeuralNetwork::~NeuralNetwork() {
    for (auto layer : layers) {
        delete layer;
    }
}

void NeuralNetwork::addLayer(NNLayer* layer) {
    this->layers.push_back(layer);
}

Matrix NeuralNetwork::forward(Matrix X) {
    Matrix Z = X;

    for (auto layer : layers) {
        Z = layer->forward(Z);
    }

    Y = Z;
    return Y;
}

void NeuralNetwork::backward(Matrix predictions, Matrix target) {
    dY.allocateMemoryIfNotAllocated(predictions.shape);
    Matrix error = bce_cost.dCost(predictions, target, dY);

    for (auto it = this->layer.rbegin(); it != this->layers.rend(); it++) {
        error = (*it)->backward(error, learning_rate);
    }

    hipDeviceSynchronize();
}

std::vector<NNLayer*> NeuralNetwork::getLayers() const {
    return layers;
}
