#include <algorithm>
#include <cassert>
#include <chrono>
#include <cstdio>
#include <functional>
#include <iomanip>
#include <iostream>
#include <random>
#include <vector>

#include <hip/hip_runtime.h>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(const char* const file, const int line)
{
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

template <class T>
float measure_performance(std::function<T(hipStream_t)> bound_function,
                          hipStream_t stream, int num_repeats = 100,
                          int num_warmups = 100)
{
    hipEvent_t start, stop;
    float time;

    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));

    for (int i{0}; i < num_warmups; ++i)
    {
        bound_function(stream);
    }

    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));

    CHECK_CUDA_ERROR(hipEventRecord(start, stream));
    for (int i{0}; i < num_repeats; ++i)
    {
        bound_function(stream);
    }
    CHECK_CUDA_ERROR(hipEventRecord(stop, stream));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    CHECK_LAST_CUDA_ERROR();
    CHECK_CUDA_ERROR(hipEventElapsedTime(&time, start, stop));
    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));

    float const latency{time / num_repeats};

    return latency;
}

constexpr size_t div_up(size_t a, size_t b) { return (a + b - 1) / b; }

template <typename T>
__global__ void transpose_read_coalesced(T* output_matrix,
                                         T const* input_matrix, size_t M,
                                         size_t N)
{
    size_t const j{threadIdx.x + blockIdx.x * blockDim.x};
    size_t const i{threadIdx.y + blockIdx.y * blockDim.y};
    size_t const from_idx{i * N + j};
    if ((i < M) && (j < N))
    {
        size_t const to_idx{j * M + i};
        output_matrix[to_idx] = input_matrix[from_idx];
    }
}

template <typename T>
__global__ void transpose_write_coalesced(T* output_matrix,
                                          T const* input_matrix, size_t M,
                                          size_t N)
{
    size_t const j{threadIdx.x + blockIdx.x * blockDim.x};
    size_t const i{threadIdx.y + blockIdx.y * blockDim.y};
    size_t const to_idx{i * M + j};
    if ((i < N) && (j < M))
    {
        size_t const from_idx{j * N + i};
        output_matrix[to_idx] = input_matrix[from_idx];
    }
}

template <typename T>
void launch_transpose_read_coalesced(T* output_matrix, T const* input_matrix,
                                     size_t M, size_t N, hipStream_t stream)
{
    constexpr size_t const warp_size{32};
    dim3 const threads_per_block{warp_size, warp_size};
    dim3 const blocks_per_grid{static_cast<unsigned int>(div_up(N, warp_size)),
                               static_cast<unsigned int>(div_up(M, warp_size))};
    transpose_read_coalesced<<<blocks_per_grid, threads_per_block, 0, stream>>>(
        output_matrix, input_matrix, M, N);
    CHECK_LAST_CUDA_ERROR();
}

template <typename T>
void launch_transpose_write_coalesced(T* output_matrix, T const* input_matrix,
                                      size_t M, size_t N, hipStream_t stream)
{
    constexpr size_t const warp_size{32};
    dim3 const threads_per_block{warp_size, warp_size};
    dim3 const blocks_per_grid{static_cast<unsigned int>(div_up(M, warp_size)),
                               static_cast<unsigned int>(div_up(N, warp_size))};
    transpose_write_coalesced<<<blocks_per_grid, threads_per_block, 0,
                                stream>>>(output_matrix, input_matrix, M, N);
    CHECK_LAST_CUDA_ERROR();
}

template <typename T, size_t BLOCK_SIZE = 32>
__global__ void transpose_read_write_coalesced(T* output_matrix,
                                               T const* input_matrix, size_t M,
                                               size_t N)
{
    // BLOCK_SIZE + 1 for avoiding the shared memory bank conflicts.
    // https://leimao.github.io/blog/CUDA-Shared-Memory-Bank/
    // Try setting it to BLOCK_SIZE instead of BLOCK_SIZE + 1 to see the
    // performance drop.
    __shared__ T buffer[BLOCK_SIZE][BLOCK_SIZE + 1];

    // Make sure blockDim.x == blockDim.y == BLOCK_SIZE

    size_t const matrix_j{threadIdx.x + blockIdx.x * blockDim.x};
    size_t const matrix_i{threadIdx.y + blockIdx.y * blockDim.y};
    size_t const matrix_from_idx{matrix_i * N + matrix_j};

    if ((matrix_i < M) && (matrix_j < N))
    {
        buffer[threadIdx.x][threadIdx.y] = input_matrix[matrix_from_idx];
    }

    // Make sure the buffer in a block is filled.
    __syncthreads();

    size_t const matrix_transposed_j{threadIdx.x + blockIdx.y * blockDim.y};
    size_t const matrix_transposed_i{threadIdx.y + blockIdx.x * blockDim.x};

    if ((matrix_transposed_i < N) && (matrix_transposed_j < M))
    {
        size_t const to_idx{matrix_transposed_i * M + matrix_transposed_j};
        output_matrix[to_idx] = buffer[threadIdx.y][threadIdx.x];
    }
}

template <typename T>
void launch_transpose_read_write_coalesced(T* output_matrix,
                                           T const* input_matrix, size_t M,
                                           size_t N, hipStream_t stream)
{
    constexpr size_t const warp_size{32};
    dim3 const threads_per_block{warp_size, warp_size};
    dim3 const blocks_per_grid{static_cast<unsigned int>(div_up(N, warp_size)),
                               static_cast<unsigned int>(div_up(M, warp_size))};
    transpose_read_write_coalesced<T, warp_size>
        <<<blocks_per_grid, threads_per_block, 0, stream>>>(output_matrix,
                                                            input_matrix, M, N);
    CHECK_LAST_CUDA_ERROR();
}

template <typename T>
bool is_equal(T const* data_1, T const* data_2, size_t size)
{
    for (size_t i{0}; i < size; ++i)
    {
        if (data_1[i] != data_2[i])
        {
            return false;
        }
    }
    return true;
}

template <typename T>
bool verify_transpose_implementation(
    std::function<void(T*, T const*, size_t, size_t, hipStream_t)>
        transpose_function,
    size_t M, size_t N)
{
    // Fixed random seed for reproducibility
    std::mt19937 gen{0};
    hipStream_t stream;
    size_t const matrix_size{M * N};
    std::vector<T> matrix(matrix_size, 0.0f);
    std::vector<T> matrix_transposed(matrix_size, 1.0f);
    std::vector<T> matrix_transposed_reference(matrix_size, 2.0f);
    std::uniform_real_distribution<T> uniform_dist(-256, 256);
    for (size_t i{0}; i < matrix_size; ++i)
    {
        matrix[i] = uniform_dist(gen);
    }
    // Create the reference transposed matrix using CPU.
    for (size_t i{0}; i < M; ++i)
    {
        for (size_t j{0}; j < N; ++j)
        {
            size_t const from_idx{i * N + j};
            size_t const to_idx{j * M + i};
            matrix_transposed_reference[to_idx] = matrix[from_idx];
        }
    }
    T* d_matrix;
    T* d_matrix_transposed;
    CHECK_CUDA_ERROR(hipMalloc(&d_matrix, matrix_size * sizeof(T)));
    CHECK_CUDA_ERROR(hipMalloc(&d_matrix_transposed, matrix_size * sizeof(T)));
    CHECK_CUDA_ERROR(hipStreamCreate(&stream));
    CHECK_CUDA_ERROR(hipMemcpy(d_matrix, matrix.data(),
                                matrix_size * sizeof(T),
                                hipMemcpyHostToDevice));
    transpose_function(d_matrix_transposed, d_matrix, M, N, stream);
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));
    CHECK_CUDA_ERROR(hipMemcpy(matrix_transposed.data(), d_matrix_transposed,
                                matrix_size * sizeof(T),
                                hipMemcpyDeviceToHost));
    bool const correctness{is_equal(matrix_transposed.data(),
                                    matrix_transposed_reference.data(),
                                    matrix_size)};
    CHECK_CUDA_ERROR(hipFree(d_matrix));
    CHECK_CUDA_ERROR(hipFree(d_matrix_transposed));
    CHECK_CUDA_ERROR(hipStreamDestroy(stream));
    return correctness;
}

template <typename T>
void profile_transpose_implementation(
    std::function<void(T*, T const*, size_t, size_t, hipStream_t)>
        transpose_function,
    size_t M, size_t N)
{
    constexpr int const num_repeats{100};
    constexpr int const num_warmups{10};
    hipStream_t stream;
    size_t const matrix_size{M * N};
    T* d_matrix;
    T* d_matrix_transposed;
    CHECK_CUDA_ERROR(hipMalloc(&d_matrix, matrix_size * sizeof(T)));
    CHECK_CUDA_ERROR(hipMalloc(&d_matrix_transposed, matrix_size * sizeof(T)));
    CHECK_CUDA_ERROR(hipStreamCreate(&stream));

    std::function<void(hipStream_t)> const transpose_function_wrapped{
        std::bind(transpose_function, d_matrix_transposed, d_matrix, M, N,
                  std::placeholders::_1)};
    float const transpose_function_latency{measure_performance(
        transpose_function_wrapped, stream, num_repeats, num_warmups)};
    std::cout << std::fixed << std::setprecision(3)
              << "Latency: " << transpose_function_latency << " ms"
              << std::endl;
    CHECK_CUDA_ERROR(hipFree(d_matrix));
    CHECK_CUDA_ERROR(hipFree(d_matrix_transposed));
    CHECK_CUDA_ERROR(hipStreamDestroy(stream));
}

int main()
{
    // Unit tests.
    for (size_t m{1}; m <= 64; ++m)
    {
        for (size_t n{1}; n <= 64; ++n)
        {
            assert(verify_transpose_implementation<float>(
                &launch_transpose_write_coalesced<float>, m, n));
            assert(verify_transpose_implementation<float>(
                &launch_transpose_read_coalesced<float>, m, n));
            assert(verify_transpose_implementation<float>(
                &launch_transpose_read_write_coalesced<float>, m, n));
        }
    }

    // M: Number of rows.
    size_t const M{12800};
    // N: Number of columns.
    size_t const N{12800};
    std::cout << M << " x " << N << " Matrix" << std::endl;
    std::cout << "Transpose Write Coalesced" << std::endl;
    profile_transpose_implementation<float>(
        &launch_transpose_write_coalesced<float>, M, N);
    std::cout << "Transpose Read Coalesced" << std::endl;
    profile_transpose_implementation<float>(
        &launch_transpose_read_coalesced<float>, M, N);
    std::cout << "Transpose Read and Write Coalesced" << std::endl;
    profile_transpose_implementation<float>(
        &launch_transpose_read_write_coalesced<float>, M, N);
}