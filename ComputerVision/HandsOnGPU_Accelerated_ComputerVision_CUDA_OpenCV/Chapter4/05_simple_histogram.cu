#include <stdio.h>
#include <hip/hip_runtime.h>

#define SIZE 1000
#define NUM_BIN 16

__global__ void histogram_without_atomic(int *d_b, int *d_a) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int item = d_a[tid];
    if (tid < SIZE) {
        d_b[item]++;
    }
}

__global__ void histogram_atomic(int *d_b, int *d_a) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int item = d_a[tid];
    if (tid < SIZE) {
        atomicAdd(&(d_b[item]), 1);
    }
}

int main() {
    int h_a[SIZE];
    for (int i = 0; i < SIZE; i++) {
        h_a[i] = i % NUM_BIN;
    }

    int h_b[NUM_BIN];
    for (int i = 0; i < NUM_BIN; i++) {
        h_b[i] = 0;
    }

    // declare GPU memory pointers
    int *d_a;
    int *d_b;

    // allocate GPU memory
    hipMalloc((void **)&d_a, SIZE * sizeof(int));
    hipMalloc((void **)&d_b, NUM_BIN * sizeof(int));

    // transfer the arrays to the GPU
    hipMemcpy(d_a, h_a, SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, NUM_BIN * sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel

    histogram_without_atomic<<<((SIZE + NUM_BIN - 1) / NUM_BIN), NUM_BIN>>>(d_b, d_a);
    // histogram_atomic << <((SIZE+NUM_BIN-1) / NUM_BIN), NUM_BIN >> >(d_b, d_a);

    // copy back the sum from GPU
    hipMemcpy(h_b, d_b, NUM_BIN * sizeof(int), hipMemcpyDeviceToHost);
    
    printf("Histogram using 16 bin without shared Memory is: \n");
    for (int i = 0; i < NUM_BIN; i++) {
        printf("bin %d: count %d\n", i, h_b[i]);
    }

    // Free GPU memory allocation
    hipFree(d_a);
    hipFree(d_b);
    return 0;
}

/**
Histogram using 16 bin without shared Memory is:
bin 0: count 1
bin 1: count 1
bin 2: count 1
bin 3: count 1
bin 4: count 1
bin 5: count 1
bin 6: count 1
bin 7: count 1
bin 8: count 1
bin 9: count 1
bin 10: count 1
bin 11: count 1
bin 12: count 1
bin 13: count 1
bin 14: count 1
bin 15: count 1
*/