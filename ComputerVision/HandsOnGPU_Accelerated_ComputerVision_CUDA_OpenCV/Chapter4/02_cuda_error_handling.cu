#include <hip/hip_runtime.h>



#include <stdio.h>
__global__ void gpuAdd(int *d_a, int *d_b, int *d_c) {
    *d_c = *d_a + *d_b;
}

int main() {
    int h_a, h_b, h_c;
    int *d_a, *d_b, *d_c;

    h_a = 1;
    h_b = 4;

    hipError_t cudaStatus;
    // Allocate GPU buffers for 3 vectors
    cudaStatus = hipMalloc((void**)&d_c, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    
    cudaStatus = hipMalloc((void**)&d_a, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&d_b, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy Input vectors from Host memory to GPU buffers
    cudaStatus = hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch Kernel on GPU with 1 Thread per element
    gpuAdd<<<1, 1>>>(d_a, d_b, d_c);

    // Check for any errors launching Kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Copy output Vector from GPU buffer to host memory
    cudaStatus = hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    printf("Passing Parameter by Reference Output: %d + %d = %d\n", h_a, h_b, h_c);
Error:
    hipFree(d_c);
    hipFree(d_a);
    hipFree(d_b);

    return 0;

}