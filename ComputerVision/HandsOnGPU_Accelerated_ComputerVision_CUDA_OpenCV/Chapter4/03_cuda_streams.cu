#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>


#define N 50000

__global__ void gpuAdd(int *d_a, int *d_b, int *d_c) {
    // Getting Thread Idx of current Kernel
    int tid = threadIdx.x + blockIdx.x * blockIdx.x;
    
    while (tid < N) {
        d_c[tid] = d_a[tid] + d_b[tid];
        tid += blockDim.x * gridDim.x;
    }
}


int main() {
    int *h_a, *h_b, *h_c;
    int *d_a0, *d_b0, *d_c0; // Device Ptr for stream 0
    int *d_a1, *d_b1, *d_c1; // Device Ptr for stream 1

    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);
    
    hipEvent_t e_start, e_stop;
    hipEventCreate(&e_start);
    hipEventCreate(&e_stop);
    hipEventRecord(e_start, 0);

    hipHostAlloc((void**)&h_a, N * 2 * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&h_b, N * 2 * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&h_c, N * 2 * sizeof(int), hipHostMallocDefault);

    // Allocate Memory
    hipMalloc((void **)&d_a0, N * sizeof(int));
    hipMalloc((void **)&d_b0, N * sizeof(int));
    hipMalloc((void **)&d_c0, N * sizeof(int));
    hipMalloc((void **)&d_a1, N * sizeof(int));
    hipMalloc((void **)&d_b1, N * sizeof(int));
    hipMalloc((void **)&d_c1, N * sizeof(int));

    for (int i = 0; i < N * 2; i++) {
        h_a[i] = 2 * i * i;
        h_b[i] = i;
    }

    hipMemcpyAsync(d_a0, h_a, N * sizeof(int), hipMemcpyHostToDevice, stream0);
    hipMemcpyAsync(d_a1, h_a + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(d_b0, h_b, N * sizeof(int), hipMemcpyHostToDevice, stream0);
    hipMemcpyAsync(d_b1, h_b + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);

    // Calling Kernel passing device ptr as params
    gpuAdd<<<512, 512, 0, stream0>>>(d_a0, d_b0, d_c0);
    gpuAdd<<<512, 512, 0, stream1>>>(d_a1, d_b1, d_c1);
    // Copy result back to host memory from device memory
    hipMemcpyAsync(h_c, d_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0);
    hipMemcpyAsync(h_c + N, d_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream0);

    hipDeviceSynchronize();
    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);
    hipEventRecord(e_stop, 0);
    hipEventSynchronize(e_stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, e_start, e_stop);
    printf("Time to add %d numbers: %3.1f ms\n", 2 * N, elapsedTime);

    int correct = 1;
    printf("Vector Addition on GPU:\n");
    for (int i = 0; i < 2 * N; i++) {
        if ((h_a[i] + h_b[i] != h_c[i])) {
            correct = 0;
        }
    }

    if (correct == 1) {
        printf("GPU has computed sum correctly");
    } else {
        printf("There is an Error in GPU Computation\n");
    }

    hipFree(d_a0);
    hipFree(d_b0);
    hipFree(d_c0);
    hipFree(d_a0);
    hipFree(d_b0);
    hipFree(d_c0);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    
    return 0;
}

/**
Time to add 100000 numbers: 2.5 ms
Vector Addition on GPU:
GPU has computed sum correctly
*/