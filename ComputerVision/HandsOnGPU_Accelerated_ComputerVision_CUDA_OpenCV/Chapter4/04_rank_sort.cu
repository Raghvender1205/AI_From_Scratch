#include <stdio.h>
#include <hip/hip_runtime.h>



#define arraySize 100
#define ThreadPerBlock 5

__global__ void addKernel(int *d_a, int *d_b) {
    int cnt = 0;
    int tid = threadIdx.x;
    int ttid = blockIdx.x * ThreadPerBlock + tid;
    int val = d_a[ttid];

    __shared__ int cache[ThreadPerBlock];
    for (int i = tid; i < arraySize; i += ThreadPerBlock) {
        cache[tid] = d_a[i];
        __syncthreads();
        for (int j = 0; j < ThreadPerBlock; j++) {
            if (val > cache[j]) {
                cnt++;
            }
        }
        __syncthreads();
    }
    d_b[cnt] = val;
}

int main() {
    int h_a[arraySize] = {5, 9, 3, 4, 8};
    int h_b[arraySize];
    int *d_a, *d_b;

    hipMalloc((void **)&d_b, arraySize * sizeof(int));
    hipMalloc((void **)&d_a, arraySize * sizeof(int));

    // Copy input vector from host memory to GPU buffers.
    hipMemcpy(d_a, h_a, arraySize * sizeof(int), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<arraySize / ThreadPerBlock, ThreadPerBlock>>>(d_a, d_b);

    hipDeviceSynchronize();
    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(h_b, d_b, arraySize * sizeof(int), hipMemcpyDeviceToHost);
    printf("The Enumeration sorted Array is: \n");
    for (int i = 0; i < arraySize; i++) {
        printf("%d\n", h_b[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    return 0;
}