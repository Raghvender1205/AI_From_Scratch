#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N	5

// Kernel function for squaring number
__global__ void gpuSquare(float *d_in, float *d_out) {
	// Getting thread index for current kernel
	int tid = threadIdx.x;	// handle the data at this index
	float temp = d_in[tid];
	d_out[tid] = temp*temp;
}

int main(void) {
	float h_in[N], h_out[N];
	// Defining Pointers for device
	float *d_in, *d_out;

	// Allocate the memory on the cpu
	hipMalloc((void**)&d_in, N * sizeof(float));
	hipMalloc((void**)&d_out, N * sizeof(float));
	
    // Initializing Array
	for (int i = 0; i < N; i++) {
		h_in[i] = i;
	}
	// Copy Array from host to device
	hipMemcpy(d_in, h_in, N * sizeof(float), hipMemcpyHostToDevice);
	// Calling square kernel with one block and N threads per block
	gpuSquare << <1, N >> >(d_in, d_out);
	// Coping result back to host from device memory
	hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);
	
    printf("Square of Number on GPU \n");
	for (int i = 0; i < N; i++) {
		printf("The square of %f is %f\n", h_in[i], h_out[i]);
	}
	// Free up memory
	hipFree(d_in);
	hipFree(d_out);
	
    return 0;
}

/**
Square of Number on GPU 
The square of 0.000000 is 0.000000
The square of 1.000000 is 1.000000
The square of 2.000000 is 4.000000
The square of 3.000000 is 9.000000
The square of 4.000000 is 16.000000
*/