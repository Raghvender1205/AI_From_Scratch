
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 5

__global__ void gpu_global_memory(int *d_a) {
    d_a[threadIdx.x] = threadIdx.x;
}

int main(int argc, char **argv) {
    // Define Host Array
    int h_a[N];
    // Define device pointer
    int *d_a;

    hipMalloc((void **)&d_a, sizeof(int) * N);
    // now copy data from host memory to device memory
    hipMemcpy((void *)d_a, (void *)h_a, sizeof(int) * N, hipMemcpyHostToDevice);
    // launch the kernel
    gpu_global_memory<<<1, N>>>(d_a);
    // copy the modified array back to the host memory
    hipMemcpy((void *)h_a, (void *)d_a, sizeof(int) * N, hipMemcpyDeviceToHost);
    
    printf("Array in Global Memory is: \n");
    for (int i = 0; i < N; i++) {
        printf("At Index: %d --> %d \n", i, h_a[i]);
    }

    return 0;
}