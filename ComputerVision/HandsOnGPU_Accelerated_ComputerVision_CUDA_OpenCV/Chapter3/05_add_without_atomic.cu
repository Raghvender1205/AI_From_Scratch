#include "hip/hip_runtime.h"
#include <stdio.h>

#define NUM_THREADS 10000
#define SIZE 10

#define BLOCK_WIDTH 100

__global__ void gpu_increment_without_atomic(int* d_a) {
    // Cal. thread id for current thread
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Each thread increments elements wrapping at SIZE variable
    tid = tid % SIZE;
    d_a[tid] += 1;
}

int main() {
    printf("%d total threads in %d blocks writing into %d array elements\n",
            NUM_THREADS, NUM_THREADS / BLOCK_WIDTH, SIZE);
    // declare and allocate host memory
    int h_a[SIZE];
    const int ARRAY_BYTES = SIZE * sizeof(int);

    // Declare and allocate GPU memory
    int* d_a;
    hipMalloc((void **)&d_a, ARRAY_BYTES);
    // Initiate GPU memory to zero
    hipMemset((void *)d_a, 0, ARRAY_BYTES);

    gpu_increment_without_atomic << <NUM_THREADS / BLOCK_WIDTH, BLOCK_WIDTH >> >(d_a);
    // Copy back the array to host memory
    hipMemcpy(h_a, d_a, ARRAY_BYTES, hipMemcpyDeviceToHost);

    printf("Number of times a particular Array index has been incremented without atomic add is: \n");
	for (int i = 0; i < SIZE; i++) {
		printf("index: %d --> %d times\n ", i, h_a[i]);
	}

	hipFree(d_a);
	return 0;
}

/**
10000 total threads in 100 blocks writing into 10 array elements
Number of times a particular Array index has been incremented without atomic add is: 
index: 0 --> 1 times
 index: 1 --> 1 times
 index: 2 --> 1 times
 index: 3 --> 1 times
 index: 4 --> 1 times
 index: 5 --> 1 times
 index: 6 --> 1 times
 index: 7 --> 1 times
 index: 8 --> 1 times
 index: 9 --> 1 times
*/