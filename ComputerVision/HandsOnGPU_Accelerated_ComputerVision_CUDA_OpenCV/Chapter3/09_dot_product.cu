#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N 1024
#define threadsPerBlock 512


__global__ void gpu_dot(float *d_a, float *d_b, float *d_c) {
	__shared__ float partial_sum[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	// Calculate index for shared memory 
	int index = threadIdx.x;
	
    // Calculate Partial Sum
	float sum = 0;
	while (tid < N) {
		sum += d_a[tid] * d_b[tid];
		tid += blockDim.x * gridDim.x;
	}

	// Store partial sum in shared memory
	partial_sum[index] = sum;

	// synchronize threads 
	__syncthreads();

	// Calculating partial sum for whole block in reduce operation
	int i = blockDim.x / 2;
	while (i != 0) {
		if (index < i)
			partial_sum[index] += partial_sum[index + i];
		__syncthreads();
		i /= 2;
	}
	// Store block partial sum in global memory
	if (index == 0)
		d_c[blockIdx.x] = partial_sum[0];
}


int main(void) {
	float *h_a, *h_b, h_c, *partial_sum;
	float *d_a, *d_b, *d_partial_sum;
	
    // Calculate total number of blocks per grid
	int block_calc = (N + threadsPerBlock - 1) / threadsPerBlock;
	int blocksPerGrid = (32 < block_calc ? 32 : block_calc);
	// Allocate memory on the host side
	h_a = (float*)malloc(N * sizeof(float));
	h_b = (float*)malloc(N * sizeof(float));
	partial_sum = (float*)malloc(blocksPerGrid * sizeof(float));

	// Allocate the memory on the device
	hipMalloc((void**)&d_a, N * sizeof(float));
	hipMalloc((void**)&d_b, N * sizeof(float));
	hipMalloc((void**)&d_partial_sum, blocksPerGrid * sizeof(float));

	// Fill the host array with data
	for (int i = 0; i<N; i++) {
		h_a[i] = i;
		h_b[i] = 2;
	}

	hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);
	// Call kernel 
	gpu_dot << <blocksPerGrid, threadsPerBlock >> >(d_a, d_b, d_partial_sum);

	// Copy the array back to host memory
	hipMemcpy(partial_sum, d_partial_sum, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);

	// Calculate final dot product on host
	h_c = 0;
	for (int i = 0; i<blocksPerGrid; i++) {
		h_c += partial_sum[i];
	}
	printf("The computed dot product is: %f\n", h_c);
#define cpu_sum(x) (x*(x+1))
	if (h_c == cpu_sum((float)(N - 1))) {
		printf("The dot product computed by GPU is correct\n");
	} else {
		printf("Error in dot product computation");
	}

	// Free memory on host and device
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_partial_sum);
	free(h_a);
	free(h_b);
	free(partial_sum);
}

/**
The computed dot product is: 1047552.000000
The dot product computed by GPU is correct
*/